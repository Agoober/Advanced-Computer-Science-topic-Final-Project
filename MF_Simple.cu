#include "hip/hip_runtime.h"
    /*
Matrix Factorization Simple
*/

#include "mm_helper.hpp"
#include "sparse_representation.hpp"
#include <iostream>

#include <getopt.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define threadNum 256

__global__ void generate_random_numbers(float* numbers, int Np) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < Np) {

        hiprandState state;

        hiprand_init(clock64(), i, 0, &state);

        numbers[i] = hiprand_uniform(&state);
    }
}


__global__ void device_mf(unsigned int* mat_row_indx, unsigned int* mat_col_id, double* mat_values, unsigned int mat_nrows, 
     double * S_mat, double i, double j, double k, double *A, double *B, double *C){

    unsigned int threadId = blockIdx.x * threadNum + threadIdx.x;
    unsigned int tx = threadIdx.x;
    double S,G,a,b,c;

    for(int t = threadId; t<threadId + 5; ++t) {
        hiprandState state;
        hiprand_init(clock64(), 0, 0, &state);

        a = hiprand(&state)%100;
        b = hiprand(&state)%100;
        c = hiprand(&state)%100;

        for (unsigned int r = 0; r < mat_nrows; ++r) {
            unsigned int row_start = mat_row_indx[r];
            unsigned int row_end = mat_row_indx[r + 1];
            //dmat_out[r * K + threadId] = 0;
            for (unsigned int p = row_start; p < row_end; ++p) {
                //unsigned int col_id = mat_col_id[p];
                double val = mat_values[p];
                S += a*i + b*j + c*k - val;
            }
        }
        S_mat[t] = S;
        A[t] = a;
        B[t] = b;
        C[t] = c;
    }
   
}



int main(int argc, char *argv[]) {

    if(argc < 2) {
        std::cerr << "usage ./exec inputfile " << std::endl;
        exit(-1);
    }

    CSR mat = read_matrix_market_to_CSR(argv[1]);
    std::cout << mat.nrows << ' ' << mat.ncols << ' ' << mat.nnz << ' ' << '\n';

    

    unsigned int* row_indx_device;
    unsigned int* col_id_device;
    double* values_device;

    hipMalloc((unsigned int**)&row_indx_device, (mat.nrows + 1) * sizeof(unsigned int));
    hipMemcpy(row_indx_device, mat.row_indx, (mat.nrows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);

    hipMalloc((unsigned int**)&col_id_device, mat.nnz * sizeof(unsigned int));
    hipMemcpy(col_id_device, mat.col_id, mat.nnz * sizeof(unsigned int), hipMemcpyHostToDevice);

    hipMalloc((double**)&values_device, mat.nnz * sizeof(double));
    hipMemcpy(values_device, mat.values, mat.nnz * sizeof(double), hipMemcpyHostToDevice);

    double *dSmat, *dA, *dB, *dC;
    hipMalloc((double**)&dSmat, 5*mat.nnz*sizeof(double) );
    hipMalloc((double**)&dA, 5*mat.nnz*sizeof(double) );
    hipMalloc((double**)&dB, 5*mat.nnz*sizeof(double) );
    hipMalloc((double**)&dC, 5*mat.nnz*sizeof(double) );

    double di, dj,dk;
    di = rand()/100;
    dj = rand()/100;
    dk = rand()/100;

    dim3 threads = dim3(threadNum, 1,1);
    dim3 blocks = dim3(mat.nnz/threadNum + 1,1,1);

    device_mf<<<blocks, threads>>>(row_indx_device, col_id_device, values_device, mat.nrows,dSmat,di, dj, dk, dA, dB, dC);
    
    
    hipDeviceSynchronize();
    double *hSmat, *hA, *hB, *hC;
  
    hipMemcpy(hSmat, dSmat, mat.nnz * 5 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hA, dA, mat.nnz * 5 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hB, dB, mat.nnz * 5 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hC, dC, mat.nnz * 5 * sizeof(double), hipMemcpyDeviceToHost);

    int tem = hSmat[0];
    double a_out, b_out, c_out;
    for (int n = 0; n<mat.nnz*5; n++) {
        if (tem>hSmat[n]) {
            tem = hSmat[n];
            a_out = hA[n];
            b_out = hB[n];
            c_out = hB[n];
        }
    }

    double result = a_out*di + b_out*dj + c_out*dk;

    std::cout<< result;


    free(mat.row_indx);
    free(mat.col_id);
    free(mat.values);
    free(hSmat);
    free(hA);
    free(hB);
    free(hC);
    hipFree(dSmat);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(row_indx_device);
    hipFree(col_id_device);
    hipFree(values_device);




    return 0;
}