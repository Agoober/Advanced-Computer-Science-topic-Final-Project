#include "hip/hip_runtime.h"
#include "svd_interface.cuh"

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include <hipblas.h>
#include <hipsolver.h>

__global__ void uxs(float* u, float* s, float* out, int m, int n, int k)
{
	// s = m*n
	// u = m*m
	// v = n*n
	// u = m*k
	const int idx = threadIdx.x;
	for (size_t i = 0; i < k; i++)
	{
		const int ele_index = idx * n + i;
		out[idx * k + i] = u[ele_index] * s[i];
	}
}

void svd_decompose(float* A, float* left, float* right, int m, int n, int k)
{
	double accum; 
	hipsolverHandle_t cusolverH; 
	hipblasHandle_t cublasH; 
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat = hipSuccess;

	const int lda = m;
	float* U, * VT, * S; 
	U = (float*)malloc(lda * m * sizeof(float));
	VT = (float*)malloc(lda * n * sizeof(float));
	S = (float*)malloc(n * sizeof(float));

	float* d_A, * d_U, * d_VT, * d_S; 
	int* devInfo; 
	float* d_work, * d_rwork; 
	float* d_W; 
	int lwork = 0;
	int info_gpu = 0; 
	const float h_one = 1;
	const float h_minus_one = -1;

	// create cusolver and cublas handle
	cusolver_status = hipsolverDnCreate(&cusolverH);
	cublas_status = hipblasCreate(&cublasH);

	// prepare memory on the device
	cudaStat = hipMalloc((void**)&d_A, sizeof(float) * lda * n);
	cudaStat = hipMalloc((void**)&d_S, sizeof(float) * n);
	cudaStat = hipMalloc((void**)&d_U, sizeof(float) * lda * m);
	cudaStat = hipMalloc((void**)&d_VT, sizeof(float) * lda * n);
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	cudaStat = hipMalloc((void**)&d_W, sizeof(float) * lda * n);
	cudaStat = hipMemcpy(d_A, A, sizeof(float) * lda * n,
		hipMemcpyHostToDevice); // copy A- >d_A

			// compute buffer size and prepare workspace
	cusolver_status = hipsolverDnSgesvd_bufferSize(cusolverH, m, n,
		&lwork);
	cudaStat = hipMalloc((void**)&d_work, sizeof(float) * lwork);

	signed char jobu = 'A'; 
	signed char jobvt = 'A'; 



	auto st = std::chrono::system_clock::now();
	std::cout << "Start to time!" << std::endl;

	printf("SVD time :sec .\n"); 

	cusolver_status = hipsolverDnSgesvd(cusolverH, jobu, jobvt,
		m, n, d_A, lda, d_S, d_U, lda, d_VT, lda, d_work, lwork,
		d_rwork, devInfo);
	cudaStat = hipDeviceSynchronize();

	auto end = std::chrono::system_clock::now();

	auto dur = std::chrono::duration_cast<std::chrono::milliseconds>(end - st);
	std::cout << "hipsolverDnSgesvd time: " << dur.count() << " ms" << std::endl;

	cudaStat = hipMemcpy(U, d_U, sizeof(float) * lda * m,
		hipMemcpyDeviceToHost); 
	cudaStat = hipMemcpy(VT, d_VT, sizeof(float) * lda * n,
		hipMemcpyDeviceToHost); 
	cudaStat = hipMemcpy(S, d_S, sizeof(float) * n,
		hipMemcpyDeviceToHost); 
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int),
		hipMemcpyDeviceToHost); 
	printf(" after gesvd : info_gpu = %d\n", info_gpu);
	
	cublas_status = hipblasSdgmm(cublasH, HIPBLAS_SIDE_LEFT, n, n,
		d_VT, lda, d_S, 1, d_W, lda); 
	cudaStat = hipMemcpy(d_A, A, sizeof(float) * lda * n,
		hipMemcpyHostToDevice); 
		
	cublas_status = hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
		m, n, n, &h_minus_one, d_U, lda, d_W, lda, &h_one, d_A, lda);

	float dR_fro = 0.0; 
	// compute the norm of the difference d_A -d_U *d_S * d_VT
	cublas_status = hipblasSnrm2(cublasH, lda * n, d_A, 1, &dR_fro);
	printf("|A - U*S*VT| = %E \n", dR_fro); // print the norm

	float* d_left, * d_right;
	cudaStat = hipMalloc((void**)&d_left, sizeof(float) * m * k);

	uxs << <1, k >> > (d_U, d_S, d_left, m, n, k);

	if (left) {
		//std::cout << "left" << std::endl;
		cudaStat = hipMemcpy(left, d_left, sizeof(float) * m * k, hipMemcpyDeviceToHost);
	}

	cudaStat = hipMemcpy(right, d_VT, sizeof(float) * k * n, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_S);
	hipFree(d_U);
	hipFree(d_VT);
	hipFree(devInfo);
	hipFree(d_work);
	hipFree(d_rwork);
	hipFree(d_W);
	hipFree(d_left);
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);
	hipDeviceReset();

}